#include "hip/hip_runtime.h"
#include "psrdada_cpp/meerkat/fbfuse/test/CoherentBeamformerTester.cuh"
#include "psrdada_cpp/meerkat/fbfuse/fbfuse_constants.hpp"
#include "psrdada_cpp/common.hpp"
#include "psrdada_cpp/cuda_utils.hpp"
#include <random>
#include <cmath>
#include <complex>

namespace psrdada_cpp {
namespace meerkat {
namespace fbfuse {
namespace test {

CoherentBeamformerTester::CoherentBeamformerTester()
    : ::testing::Test()
    , _stream(0)
{

}

CoherentBeamformerTester::~CoherentBeamformerTester()
{


}

void CoherentBeamformerTester::SetUp()
{
    CUDA_ERROR_CHECK(hipStreamCreate(&_stream));
}

void CoherentBeamformerTester::TearDown()
{
    CUDA_ERROR_CHECK(hipStreamDestroy(_stream));
}

void CoherentBeamformerTester::beamformer_c_reference(
    HostVoltageVectorType const& ftpa_voltages,
    HostWeightsVectorType const& fbpa_weights,
    HostPowerVectorType& tbtf_powers,
    int nchannels,
    int tscrunch,
    int fscrunch,
    int nsamples,
    int nbeams,
    int nantennas,
    int npol,
    float const* scales,
    float const* offsets)
{
    float xx,yy,xy,yx;
    double power_sum = 0.0;
    double power_sq_sum = 0.0;
    std::size_t count = 0;
    for (int channel_idx = 0; channel_idx < nchannels; channel_idx += fscrunch)
    {
        BOOST_LOG_TRIVIAL(debug) << "Beamformer C reference: "
        << static_cast<int>(100.0f * (channel_idx + 1.0f) / nchannels)
        << "% complete";
        for (int sample_idx = 0; sample_idx < nsamples; sample_idx+=tscrunch)
        {
            for (int beam_idx = 0; beam_idx < nbeams; ++beam_idx)
            {
                float power = 0.0f;

                for (int sub_channel_idx = channel_idx;
                   sub_channel_idx < channel_idx + fscrunch;
                   ++sub_channel_idx)
                {
                    for (int sample_offset = 0; sample_offset < tscrunch; ++sample_offset)
                    {
                        for (int pol_idx = 0; pol_idx < npol; ++pol_idx)
                        {
                            float2 accumulator = {0,0};
                            for (int antenna_idx = 0; antenna_idx < nantennas; ++antenna_idx)
                            {
                                int ftpa_voltages_idx = nantennas * npol * nsamples * sub_channel_idx
                                + nantennas * npol * (sample_idx + sample_offset)
                                + nantennas * pol_idx
                                + antenna_idx;
                                char2 datum = ftpa_voltages[ftpa_voltages_idx];

                                int fbpa_weights_idx = nantennas * nbeams * sub_channel_idx
                                + nantennas * beam_idx
                                + antenna_idx;
                                char2 weight = fbpa_weights[fbpa_weights_idx];

                                xx = datum.x * weight.x;
                                yy = datum.y * weight.y;
                                xy = datum.x * weight.y;
                                yx = datum.y * weight.x;
                                accumulator.x += xx - yy;
                                accumulator.y += xy + yx;
                            }
                            float r = accumulator.x;
                            float i = accumulator.y;
                            power += r*r + i*i;
                        }
                    }
                }
                int tf_size = FBFUSE_CB_NSAMPLES_PER_HEAP * nchannels/fscrunch;
                int btf_size = nbeams * tf_size;
                int output_sample_idx = sample_idx / tscrunch;
                int tbtf_powers_idx = (output_sample_idx / FBFUSE_CB_NSAMPLES_PER_HEAP * btf_size
                    + beam_idx * tf_size
                    + (output_sample_idx % FBFUSE_CB_NSAMPLES_PER_HEAP) * nchannels/fscrunch
                    + channel_idx/fscrunch);
                power_sum += power;
                power_sq_sum += power * power;
                ++count;	
    		float powerf32 = ((power - offsets[channel_idx/fscrunch])/scales[channel_idx/fscrunch]);
		tbtf_powers[tbtf_powers_idx] = (int8_t) fmaxf(-127.0f, fminf(127.0f, powerf32));	
            }
        }
    }
    double power_mean = power_sum / count;
    BOOST_LOG_TRIVIAL(debug) << "Average power level: " << power_mean;
    BOOST_LOG_TRIVIAL(debug) << "Power variance: " << power_sq_sum / count - power_mean * power_mean;
}

void CoherentBeamformerTester::compare_against_host(
    DeviceVoltageVectorType const& ftpa_voltages_gpu,
    DeviceWeightsVectorType const& fbpa_weights_gpu,
    DeviceScalingVectorType const& scales_gpu,
    DeviceScalingVectorType const& offsets_gpu,
    DevicePowerVectorType& btf_powers_gpu,
    int nsamples)
{
    HostVoltageVectorType ftpa_voltages_host = ftpa_voltages_gpu;
    HostWeightsVectorType fbpa_weights_host = fbpa_weights_gpu;
    HostPowerVectorType btf_powers_cuda = btf_powers_gpu;
    HostPowerVectorType btf_powers_host(btf_powers_gpu.size());


    HostScalingVectorType scales = scales_gpu;
    HostScalingVectorType offsets = offsets_gpu;

    beamformer_c_reference(ftpa_voltages_host,
        fbpa_weights_host,
        btf_powers_host,
        _config.nchans(),
        _config.cb_tscrunch(),
        _config.cb_fscrunch(),
        nsamples,
        _config.cb_nbeams(),
        _config.cb_nantennas(),
        _config.npol(),
        thrust::raw_pointer_cast(scales.data()),
        thrust::raw_pointer_cast(offsets.data()));
    for (int ii = 0; ii < btf_powers_host.size(); ++ii)
    {
        EXPECT_NEAR(btf_powers_host[ii], btf_powers_cuda[ii], 1);
    }
}

TEST_F(CoherentBeamformerTester, representative_noise_test)
{
    const float input_level = 32.0f;
    const double pi = std::acos(-1);
    _config.output_level(input_level);   
    
    float scale = std::pow(127.0f * input_level * std::sqrt(static_cast<float>(_config.cb_nantennas())), 2);
    float dof = 2 * _config.cb_tscrunch() * _config.cb_fscrunch() * _config.npol();
    float offset_val = (scale * dof);
    float scale_val = (scale * std::sqrt(2 * dof) / _config.output_level());

    /*
    printf("Nantennas: %d, tscrunch: %d, fscrunch: %d, npol: %d, Output level: %f, Input level: %f, Scale val: %f, Offset val: %f\n", 
           _config.cb_nantennas(), _config.cb_tscrunch(), _config.cb_fscrunch(), _config.npol(), 
	   _config.output_level(), input_level, scale_val, offset_val);
    */


    DeviceScalingVectorType scales(_config.nchans() / _config.cb_fscrunch(), scale_val);
    DeviceScalingVectorType offsets(_config.nchans() / _config.cb_fscrunch(), offset_val);

    std::default_random_engine generator;
    std::normal_distribution<float> normal_dist(0.0, input_level);
    std::uniform_real_distribution<float> uniform_dist(0.0, 2*pi);

    CoherentBeamformer coherent_beamformer(_config);
    
    std::size_t ntimestamps = max(1L, FBFUSE_CB_PACKET_SIZE/(_config.nchans()/_config.cb_fscrunch())/(_config.nsamples_per_heap()/_config.cb_tscrunch()));
    ntimestamps = max(ntimestamps, FBFUSE_CB_NSAMPLES_PER_BLOCK / _config.nsamples_per_heap());
    printf("Using %ld timestamps\n",ntimestamps);
    
    std::size_t input_size = (ntimestamps * _config.cb_nantennas()
        * _config.nchans() * _config.nsamples_per_heap() * _config.npol());
    int nsamples = _config.nsamples_per_heap() * ntimestamps;
    
    std::size_t weights_size = _config.cb_nantennas() * _config.nchans() * _config.cb_nbeams();
    
    HostVoltageVectorType ftpa_voltages_host(input_size);
    for (int ii = 0; ii < ftpa_voltages_host.size(); ++ii)
    {
        ftpa_voltages_host[ii].x = static_cast<int8_t>(std::lround(normal_dist(generator)));
        ftpa_voltages_host[ii].y = static_cast<int8_t>(std::lround(normal_dist(generator)));
    }
    
    HostWeightsVectorType fbpa_weights_host(weights_size);
    for (int ii = 0; ii < fbpa_weights_host.size(); ++ii)
    {
        // Build complex weight as C * exp(i * theta).
        std::complex<double> val = 127.0f * std::exp(std::complex<float>(0.0f, uniform_dist(generator)));
        fbpa_weights_host[ii].x = static_cast<int8_t>(std::lround(val.real()));
        fbpa_weights_host[ii].y = static_cast<int8_t>(std::lround(val.imag()));
    }

    DeviceVoltageVectorType ftpa_voltages_gpu = ftpa_voltages_host;
    DeviceWeightsVectorType fbpa_weights_gpu = fbpa_weights_host;
    DevicePowerVectorType btf_powers_gpu;
    
    coherent_beamformer.beamform(ftpa_voltages_gpu, fbpa_weights_gpu, scales, offsets, btf_powers_gpu, _stream);
    compare_against_host(ftpa_voltages_gpu, fbpa_weights_gpu, scales, offsets, btf_powers_gpu, nsamples);
}

} //namespace test
} //namespace fbfuse
} //namespace meerkat
} //namespace psrdada_cpp

